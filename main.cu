#include "hip/hip_runtime.h"
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "libs/includes.hpp"

__device__ Viewport vp;
__device__ int pw, ph, mt;

void load_image(int &w, int &h, color *&darr, std::string path) {
    color *arr;
    parse_image(w, h, arr, path);
    hipMalloc(&darr, w * h * sizeof(color));
    hipMemcpy(darr, arr, w * h * sizeof(color), hipMemcpyHostToDevice);
}

__global__ void init(int imw, int imh, color *imarr) {
    vp = Viewport(
        pw,                     // pixel width
        ph,                     // pixel height
        rad(20),                // field of view (rad)
        500,                    // samples
        0.5,                    // x-offset for sampling
        0.5,                    // y-offset for sampling
        10,                     // max depth
        point3(8, 2, 3),        // lookfrom
        point3(0.5, 0.5, 0.5),  // lookat
        vec3(0, 1, 0),          // dir up(int)(darr[0].x + 2)
        rad(0),                 // defocus angle
        1000                    // max fig cnt
    );

    auto ground_material = new Lamberitan(new Solid(color(1, 1, 1)));
    vp.world.add(new Parallelepiped(point3(-1, -10, -1), point3(9, -10, -1), point3(9, -10, 4), point3(-1, -10, 4), point3(-1, 3, -1), point3(9, 3, -1), point3(9, 3, 4), point3(-1, 3, 4)), ground_material);

    auto light_material = new Lamp(color(1, 1, 1));
    vp.world.add(new Parallelepiped(point3(-1, 0.5, 1.5), point3(-0.5, 0.5, 1.5), point3(-0.5, 0.5, 2.5), point3(-1, 0.5, 2.5), point3(-1, 1.5, 1.5), point3(-0.5, 1.5, 1.5), point3(-0.5, 1.5, 2.5), point3(-1, 1.5, 2.5)), light_material);

    auto par_material = new Lamberitan(new Solid(color(5.0 / 255, 107.0 / 255, 0)));
    for (double i = 0; i < 5.3; i += 0.5) {
        for (double j = 0; j < 5.3; j += 0.5) {
            double h = random_double(-0.5, 0.5);
            vp.world.add(new Parallelepiped(point3(i, -10, j), point3(i + 1, -10, j), point3(i + 1, -10, j + 1), point3(i, -10, j + 1), point3(i, h, j), point3(i + 1, h, j), point3(i + 1, h, j + 1), point3(i, h, j + 1)), par_material);
        }
    }

    auto earth_material = new Lamberitan(new Imtext(imw, imh, imarr));
    vp.world.add(new Sphere(point3(0.5, 1, 0.5), 0.5), earth_material);
}

__global__ void render(color *arr) {
    int i = blockIdx.x * mt + threadIdx.x;
    if (i >= pw * ph) return;
    arr[i] = prepare_color(vp.get_pixel_color(i % pw, i / pw));
}

int main() {
    int pixel_width = 1366, pixel_height = 768, max_thread = 512;
    hipMemcpyToSymbol(HIP_SYMBOL(pw), &pixel_width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ph), &pixel_height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(mt), &max_thread, sizeof(int));

    int imw, imh; color *imarr;
    load_image(imw, imh, imarr, "earthmap.ppm");
    
    init<<<1, 1>>>(imw, imh, imarr);
    hipDeviceSynchronize();

    int size = pixel_height * pixel_width;

    color *phost = (color*)malloc(size * sizeof(color)), *pdevice;
    hipMalloc(&pdevice, size * sizeof(color));
    
    render<<<(size + max_thread - 1) / max_thread, max_thread>>>(pdevice);
    hipMemcpy(phost, pdevice, size * sizeof(color), hipMemcpyDeviceToHost);

    std::cout << "P3\n" << pixel_width << ' ' << pixel_height << "\n255\n";
    for (int i = 0; i < size; ++i) {
        auto pixel_formed = phost[i];
        std::cout << std::round(pixel_formed.x) << ' ' << std::round(pixel_formed.y) << ' ' << std::round(pixel_formed.z) << '\n';
    }
    std::cout << std::endl;
}