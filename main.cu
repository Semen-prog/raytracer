#include "hip/hip_runtime.h"
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "libs/includes.hpp"

__device__ Viewport vp;
__device__ int pw, ph;

__global__ void init() {
    vp = Viewport(
        pw,               // pixel width
        ph,               // pixel height
        rad(20),          // field of view (rad)
        50,               // samples per pixel
        0.5,              // x-offset for sampling
        0.5,              // y-offset for sampling
        50,               // max depth
        point3(13, 2, 3), // lookfrom
        point3(0, 0, 0),  // lookat
        vec3(0, 1, 0),    // dir up
        rad(0),           // defocus angle
        1000               // max fig cnt
    );

    auto ground_material = new Lamberitan(color(0.5, 0.5, 0.5));
    vp.world.add(new Sphere(point3(0,-1000,0), 1000), ground_material);

    for (int a = -11; a < 11; a += 1) {
        for (int b = -11; b < 11; b += 1) {
            auto choose_mat = random_01();
            point3 center(a + 0.9 * random_01(), 0.2, b + 0.9 * random_01());

            if ((center - point3(4, 0.2, 0)).len() > 0.9) {
                Style *sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = random_color(0, 1) * random_color(0, 1);
                    sphere_material = new Lamberitan(albedo);
                    vp.world.add(new Sphere(center, 0.2, vec3(0, random_double(0, 0.2), 0)), sphere_material);
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = random_color(0.5, 1);
                    auto fuzz = random_double(0, 0.5);
                    sphere_material = new Metal(albedo, fuzz);
                    vp.world.add(new Sphere(center, 0.2, vec3(0, random_double(0, 0.2), 0)), sphere_material);
                } else {
                    // glass
                    sphere_material = new Dielectric(1.5);
                    vp.world.add(new Sphere(center, 0.2, vec3(0, random_double(0, 0.2), 0)), sphere_material);
                }
            }
        }
    }

    auto material1 = new Dielectric(1.5);
    vp.world.add(new Sphere(point3(0, 1, 0), 1.0), material1);

    auto material2 = new Lamberitan(color(0.4, 0.2, 0.1));
    vp.world.add(new Sphere(point3(-4, 1, 0), 1.0), material2);

    auto material3 = new Metal(color(0.7, 0.6, 0.5), 0.0);
    vp.world.add(new Sphere(point3(4, 1, 0), 1.0), material3);
}

__global__ void render(color *arr) {
    int i = blockIdx.x;
    arr[i] = prepare_color(vp.get_pixel_color(i % pw, i / pw));
}

int main() {
    int pixel_width = 1366, pixel_height = 768;
    hipMemcpyToSymbol(HIP_SYMBOL(pw), &pixel_width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ph), &pixel_height, sizeof(int));
    
    init<<<1, 1>>>();
    hipDeviceSynchronize();

    color *phost = (color*)malloc(pixel_height * pixel_width * sizeof(color)), *pdevice;
    hipMalloc(&pdevice, pixel_height * pixel_width * sizeof(color));
    
    render<<<pixel_height * pixel_width, 1>>>(pdevice);
    hipMemcpy(phost, pdevice, pixel_height * pixel_width * sizeof(color), hipMemcpyDeviceToHost);

    std::cout << "P3\n" << pixel_width << ' ' << pixel_height << "\n255\n";
    for (int i = 0; i < pixel_height * pixel_width; ++i) {
        auto pixel_formed = phost[i];
        std::cout << std::round(pixel_formed.x) << ' ' << std::round(pixel_formed.y) << ' ' << std::round(pixel_formed.z) << '\n';
    }
    std::cout << std::endl;
}