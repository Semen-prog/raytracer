#include "hip/hip_runtime.h"
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "libs/includes.hpp"

__device__ Viewport vp;
__device__ int pw, ph, mt;

void load_image(int &w, int &h, color *&darr, std::string path) {
    color *arr;
    parse(w, h, arr, path);
    hipMalloc(&darr, w * h * sizeof(color));
    hipMemcpy(darr, arr, w * h * sizeof(color), hipMemcpyHostToDevice);
}

__global__ void init(int imw, int imh, color *darr) {
    vp = Viewport(
        pw,               // pixel width
        ph,               // pixel height
        rad(20),          // field of view (rad)
        50,               // samples per pixel
        0.5,              // x-offset for sampling
        0.5,              // y-offset for sampling
        50,               // max depth
        point3(13, 2, 3), // lookfrom
        point3(0, 0, 0),  // lookat
        vec3(0, 1, 0),    // dir up(int)(darr[0].x + 2)
        rad(0),           // defocus angle
        1000               // max fig cnt
    );

    auto ground_material = new Lamberitan(new Solid(color(0.5, 0.5, 0.5)));
    vp.world.add(new Sphere(point3(0,-1000,0), 1000), ground_material);

    for (int a = -11; a < 11; a += 1) {
        for (int b = -11; b < 11; b += 1) {
            auto choose_mat = random_01();
            point3 center(a + 0.9 * random_01(), 0.2, b + 0.9 * random_01());

            if ((center - point3(4, 0.2, 0)).len() > 0.9) {
                Style *sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = random_color(0, 1) * random_color(0, 1);
                    sphere_material = new Lamberitan(new Solid(albedo));
                    vp.world.add(new Sphere(center, 0.2), sphere_material);
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = random_color(0.5, 1);
                    auto fuzz = random_double(0, 0.5);
                    sphere_material = new Metal(fuzz, new Solid(albedo));
                    vp.world.add(new Sphere(center, 0.2), sphere_material);
                } else {
                    // glass
                    sphere_material = new Dielectric(1.5);
                    vp.world.add(new Sphere(center, 0.2), sphere_material);
                }
            }
        }
    }

    auto material1 = new Dielectric(1.5);
    auto material2 = new Lamberitan(new Imtext(imw, imh, darr));
    auto material3 = new Metal(0.0, new Checker(2.0, color(0, 1, 0), color(0.7, 0.6, 0.5)));

    vp.world.add(new Sphere(point3(0, 1, 0), 1.0), material1);
    vp.world.add(new Sphere(point3(-4, 1, 0), 1.0), material3);
    vp.world.add(new Sphere(point3(4, 1, 0), 1.0), material2);
}

__global__ void render(color *arr) {
    int i = blockIdx.x * mt + threadIdx.x;
    if (i >= pw * ph) return;
    arr[i] = prepare_color(vp.get_pixel_color(i % pw, i / pw));
}

int main() {
    int pixel_width = 1366, pixel_height = 768, max_thread = 512;
    hipMemcpyToSymbol(HIP_SYMBOL(pw), &pixel_width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ph), &pixel_height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(mt), &max_thread, sizeof(int));

    color *darr; int imw, imh;
    load_image(imw, imh, darr, "earthmap.ppm");
    
    init<<<1, 1>>>(imw, imh, darr);
    hipDeviceSynchronize();

    int size = pixel_height * pixel_width;

    color *phost = (color*)malloc(size * sizeof(color)), *pdevice;
    hipMalloc(&pdevice, size * sizeof(color));
    
    render<<<(size + max_thread - 1) / max_thread, max_thread>>>(pdevice);
    hipMemcpy(phost, pdevice, size * sizeof(color), hipMemcpyDeviceToHost);

    std::cout << "P3\n" << pixel_width << ' ' << pixel_height << "\n255\n";
    for (int i = 0; i < size; ++i) {
        auto pixel_formed = phost[i];
        std::cout << std::round(pixel_formed.x) << ' ' << std::round(pixel_formed.y) << ' ' << std::round(pixel_formed.z) << '\n';
    }
    std::cout << std::endl;
}