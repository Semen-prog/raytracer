#include "hip/hip_runtime.h"
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "libs/includes.hpp"

__device__ Viewport vp;
__device__ int pw, ph, mt;

void load_image(int &w, int &h, color *&darr, std::string path) {
    color *arr;
    parse_image(w, h, arr, path);
    hipMalloc(&darr, w * h * sizeof(color));
    hipMemcpy(darr, arr, w * h * sizeof(color), hipMemcpyHostToDevice);
}

__global__ void init() {
    vp = Viewport(
        pw,               // pixel width
        ph,               // pixel height
        rad(20),          // field of view (rad)
        500,               // samples per pixel
        0.5,              // x-offset for sampling
        0.5,              // y-offset for sampling
        10,               // max depth
        point3(8, 2, 3), // lookfrom
        point3(0.5, 0.5, 0.5),  // lookat
        vec3(0, 1, 0),    // dir up(int)(darr[0].x + 2)
        rad(0),           // defocus angle
        1000               // max fig cnt
    );

    auto ground_material = new Lamberitan(new Solid(color(0.5, 0.5, 0.5)));
    vp.world.add(new Parallelepiped(point3(-1, 0, -1), point3(9, 0, -1), point3(9, 0, 4), point3(-1, 0, 4), point3(-1, 3, -1), point3(9, 3, -1), point3(9, 3, 4), point3(-1, 3, 4)), ground_material);

    auto rnd_material = new Metal(0, new Solid(color(0.3, 1, 0.3)));
    auto par_material = new Metal(0, new Solid(color(1, 0, 0)));
    vp.world.add(new Sphere(point3(2.5, 0.5, 0.5), 0.5), rnd_material);
    vp.world.add(new Parallelepiped(point3(0, 0, 0), point3(1, 0, 0), point3(1, 0, 1), point3(0, 0, 1), point3(0, 1, 0), point3(1, 1, 0), point3(1, 1, 1), point3(0, 1, 1)), par_material);
}

__global__ void render(color *arr) {
    int i = blockIdx.x * mt + threadIdx.x;
    if (i >= pw * ph) return;
    arr[i] = prepare_color(vp.get_pixel_color(i % pw, i / pw));
}

int main() {
    int pixel_width = 1366, pixel_height = 768, max_thread = 512;
    hipMemcpyToSymbol(HIP_SYMBOL(pw), &pixel_width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ph), &pixel_height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(mt), &max_thread, sizeof(int));
    
    init<<<1, 1>>>();
    hipDeviceSynchronize();

    int size = pixel_height * pixel_width;

    color *phost = (color*)malloc(size * sizeof(color)), *pdevice;
    hipMalloc(&pdevice, size * sizeof(color));
    
    render<<<(size + max_thread - 1) / max_thread, max_thread>>>(pdevice);
    hipMemcpy(phost, pdevice, size * sizeof(color), hipMemcpyDeviceToHost);

    std::cout << "P3\n" << pixel_width << ' ' << pixel_height << "\n255\n";
    for (int i = 0; i < size; ++i) {
        auto pixel_formed = phost[i];
        std::cout << std::round(pixel_formed.x) << ' ' << std::round(pixel_formed.y) << ' ' << std::round(pixel_formed.z) << '\n';
    }
    std::cout << std::endl;
}